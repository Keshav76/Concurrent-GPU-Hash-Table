#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <stdint.h>
#include "../tools/gputimer.h"
#include "../data/data_layout.cuh"
#include "../core/dynamic_cuckoo.cuh"
#include <chrono>
namespace ch = cuckoo_helpers;
using namespace std;
using std::chrono::duration_cast;
using HR = std::chrono::high_resolution_clock;
using HRTimer = HR::time_point;
using std::chrono::microseconds;
using std::chrono::milliseconds;
class DynamicTest {
public:
    using key_t = DataLayout<>::key_t;
    using value_t = DataLayout<>::value_t;
    static constexpr uint32_t val_lens = DataLayout<>::val_lens;

    int r = 2;
    int batch_size = 100000;  //smaller batch size: 2e4 4e4 6e4 8e4 10e4
    double lower_bound = 0.5;  //lower bound: 0.3 0.35 0.4 0.45 0.5
    double upper_bound = 0.85; //upper bound: 0.7 0.75 0.8 0.85 0.9
    int pool_len = 0;
    key_t *keys_pool_d;          // For insert keys
    key_t *search_keys_pool_d;   // For search keys
    key_t *delete_keys_pool_d;   // For delete keys
    value_t *value_pool_d, *check_pool_d;
    double init_fill_factor = 0.85;
    static key_t *read_data(char *file_name, int data_len) {
        FILE *fid;
        fid = fopen(file_name, "rb");
        key_t *pos = (key_t *) malloc(sizeof(key_t) * data_len);
        if (fid == NULL) {
            printf("file not found.\n");
            return pos;
        }
        fread(pos, sizeof(unsigned int), data_len, fid);
        fclose(fid);
        return pos;
    }

    void batch_check(value_t *check_pool_d, int32_t single_batch_size, uint32_t offset) {
        uint32_t error_cnt = 0;
        value_t *check_pool_h = new value_t[single_batch_size];
        hipMemcpy(check_pool_h, check_pool_d + offset, sizeof(value_t) * single_batch_size, hipMemcpyDeviceToHost);
        for (int i = 0; i < single_batch_size; i++) {
            for(int j = 0; j < val_lens; j++){
                if(check_pool_h[i].data[j] != i + 5 + offset){
                    ++error_cnt;
                    break;
                }
            }
        }
        if (error_cnt != 0) {
            printf("num error:%d \n", error_cnt);
        } else {
            printf("batch check ok\n");
        }
        delete[] check_pool_h;
    }

    void batch_test() {
        DynamicCuckoo<512, 512> dy_cuckoo((uint32_t)batch_size * 10 / init_fill_factor, batch_size, lower_bound, upper_bound);
        int32_t batch_num = pool_len / batch_size;
        printf("pool_len:::::::::::::::: %d\n", pool_len);
        int32_t batch_round = batch_num / 10;
        HRTimer start, end;
        // for (int repeat = 0; repeat < 10; repeat++) {
            double insert_time =0, search_time = 0, delete_time = 0;
            //for (int32_t batch_round_ptr = 0; batch_round_ptr < batch_round; ++batch_round_ptr) {
              //  int batch_ptr = batch_round_ptr * 10;
                start = HR::now();
                //for (int j = 0; j < 10; j++) {
                    dy_cuckoo.batch_insert(keys_pool_d,  value_pool_d , pool_len);
			hipDeviceSynchronize();
                //}
                end = HR::now();
                insert_time = duration_cast<milliseconds>(end - start).count();
                start = HR::now();
                //for (int j = 0; j < 10; j++) {
                    dy_cuckoo.batch_search(search_keys_pool_d,check_pool_d ,pool_len);
			hipDeviceSynchronize();
                //}
                end = HR::now();
                search_time = duration_cast<milliseconds>(end - start).count();
                start = HR::now();
                //for (int j = 0; j < 10; j++) {
                    dy_cuckoo.batch_delete(delete_keys_pool_d, nullptr,pool_len);
			hipDeviceSynchronize();
                //}
                end = HR::now();
                delete_time = duration_cast<milliseconds>(end - start).count();
                
            //}
            double insert_throughput = (pool_len) / (insert_time  * 1e-3);
            double search_throughput = (pool_len) / (search_time * 1e-3);
            double delete_throughput = (pool_len) / (delete_time  * 1e-3);
            
            printf("Insert Time = %.3lf ms | Throughput = %.2lf M ops/sec\n", 
                   insert_time / batch_round, insert_throughput / 1e6);
            printf("Search Time = %.3lf ms | Throughput = %.2lf M ops/sec\n", 
                search_time / batch_round, search_throughput / 1e6);
            printf("Delete Time = %.3lf micro sec | Throughput = %.2lf M ops/sec\n", 
                delete_time / batch_round, delete_throughput / 1e6);
        // }

    }
};


int main(int argc, char** argv) {
    using test_t = DynamicTest;

    if (argc < 9)
    {
        cout << "Usage: " << argv[0] << " insert_file search_file delete_file pool_len r batch_size lower_bound upper_bound init_fill_factor\n";
        cout << "para error\n" << endl;
        return -1;
    }

    test_t dy_test;
    char* insert_file_name = argv[1];
    char* search_file_name = argv[2];
    char* delete_file_name = argv[3];
    int pool_len = atoi(argv[4]);
    dy_test.pool_len = pool_len;
    dy_test.r = atoi(argv[5]);
    //dy_test.batch_size = atoi(argv[6]) / 10;
    dy_test.lower_bound = atof(argv[6]);
    dy_test.upper_bound = atof(argv[7]);
    dy_test.init_fill_factor = atof(argv[8]);

    // Read keys from three different files
    test_t::key_t* insert_keys_h = test_t::read_data(insert_file_name, pool_len);

    test_t::key_t* search_keys_h = test_t::read_data(search_file_name, pool_len);

    test_t::key_t* delete_keys_h = test_t::read_data(delete_file_name, pool_len);


    test_t::value_t *values_h = new test_t::value_t [pool_len], *check_h = new test_t::value_t [pool_len];
    for(int i = 0; i < pool_len; i++){
        for(int j = 0; j < DataLayout<>::val_lens; j++){
            values_h[i].data[j] = i + 5;
            check_h[i].data[j] = 0;
        }
    }

    // Allocate and copy all three key sets to device
    hipMalloc((void**)&(dy_test.keys_pool_d), sizeof(test_t::key_t) * pool_len);
    hipMalloc((void**)&(dy_test.search_keys_pool_d), sizeof(test_t::key_t) * pool_len);
    hipMalloc((void**)&(dy_test.delete_keys_pool_d), sizeof(test_t::key_t) * pool_len);
    hipMalloc((void**)&(dy_test.value_pool_d), sizeof(test_t::value_t) * pool_len);
    hipMalloc((void**)&(dy_test.check_pool_d), sizeof(test_t::value_t) * pool_len);

    hipMemcpy(dy_test.keys_pool_d, insert_keys_h, sizeof(test_t::key_t) * pool_len, hipMemcpyHostToDevice);
    hipMemcpy(dy_test.search_keys_pool_d, search_keys_h, sizeof(test_t::key_t) * pool_len, hipMemcpyHostToDevice);
    hipMemcpy(dy_test.delete_keys_pool_d, delete_keys_h, sizeof(test_t::key_t) * pool_len, hipMemcpyHostToDevice);
    hipMemcpy(dy_test.value_pool_d, values_h, sizeof(test_t::value_t) * pool_len, hipMemcpyHostToDevice);

    dy_test.batch_test();

    delete []insert_keys_h;
    delete []search_keys_h;
    delete []delete_keys_h;
    delete []values_h;
    delete []check_h;
    return 0;
}
